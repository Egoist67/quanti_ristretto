#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "ristretto/batch_norm_ristretto_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void BatchNormRistrettoLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (this->phase_ == TEST) { /////////Quantize/////////
    for (int i = 0; i < bottom.size(); ++i) {
      //std::cout << "bn_in_quant befor: " << bottom[0]->cpu_data()[1] << " " << bottom[0]->cpu_data()[2] << std::endl;
      this->QuantizeLayerInputs_gpu(bottom[i]->mutable_cpu_data(),
          bottom[i]->count());
      //std::cout << "bn_in_quant after: " << bottom[0]->cpu_data()[1] << " " << bottom[0]->cpu_data()[2] << std::endl;
    }
  }
	
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int num = bottom[0]->shape(0);
  int spatial_dim = bottom[0]->count()/(this->channels_*bottom[0]->shape(0));

  if (bottom[0] != top[0]) {
    caffe_copy(bottom[0]->count(), bottom_data, top_data);
  }


  if (this->use_global_stats_) {
    // use the stored mean/variance estimates.
    const Dtype scale_factor = this->blobs_[2]->cpu_data()[0] == 0 ?
        0 : 1 / this->blobs_[2]->cpu_data()[0];
    caffe_gpu_scale(this->variance_.count(), scale_factor,
        this->blobs_[0]->gpu_data(), this->mean_.mutable_gpu_data());
    caffe_gpu_scale(this->variance_.count(), scale_factor,
        this->blobs_[1]->gpu_data(), this->variance_.mutable_gpu_data());

    caffe_copy(this->mean_.count(), this->mean_.cpu_data(), this->weights_quantized_mean_[0]->mutable_cpu_data());
    caffe_copy(this->variance_.count(), this->variance_.cpu_data(), this->weights_quantized_variance_[0]->mutable_cpu_data());
    int rounding = this->phase_ == TEST ? this->rounding_ :
                  QuantizationParameter_Rounding_STOCHASTIC;
    this->QuantizeWeights_gpu(this->weights_quantized_mean_, rounding, false);
    //this->QuantizeWeights_gpu(this->weights_quantized_variance_, rounding, false);
  } else {
    // compute mean
    caffe_gpu_gemv<Dtype>(CblasNoTrans, this->channels_ * num, spatial_dim,
        1. / (num * spatial_dim), bottom_data,
        this->spatial_sum_multiplier_.gpu_data(), 0.,
        this->num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num, this->channels_, 1.,
        this->num_by_chans_.gpu_data(), this->batch_sum_multiplier_.gpu_data(), 0.,
        this->mean_.mutable_gpu_data());
  }

  if (this->use_global_stats_) {
      // subtract mean
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
          this->batch_sum_multiplier_.gpu_data(), this->weights_quantized_mean_[0]->gpu_data(), 0.,
          this->num_by_chans_.mutable_gpu_data());
  } else {
      // subtract mean
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
          this->batch_sum_multiplier_.gpu_data(), this->mean_.gpu_data(), 0.,
          this->num_by_chans_.mutable_gpu_data());
  }
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->channels_ * num,
      spatial_dim, 1, -1, this->num_by_chans_.gpu_data(),
      this->spatial_sum_multiplier_.gpu_data(), 1., top_data);

  if (!this->use_global_stats_) {
    // compute variance using var(X) = E((X-EX)^2)
    caffe_gpu_powx(top[0]->count(), top_data, Dtype(2),
        this->temp_.mutable_gpu_data());  // (X-EX)^2
    caffe_gpu_gemv<Dtype>(CblasNoTrans, this->channels_ * num, spatial_dim,
        1. / (num * spatial_dim), this->temp_.gpu_data(),
        this->spatial_sum_multiplier_.gpu_data(), 0.,
        this->num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num, this->channels_, 1.,
        this->num_by_chans_.gpu_data(), this->batch_sum_multiplier_.gpu_data(), 0.,
        this->variance_.mutable_gpu_data());  // E((X_EX)^2)

    // compute and save moving average
    this->blobs_[2]->mutable_cpu_data()[0] *= this->moving_average_fraction_;
    this->blobs_[2]->mutable_cpu_data()[0] += 1;
    caffe_gpu_axpby(this->mean_.count(), Dtype(1), this->mean_.gpu_data(),
        this->moving_average_fraction_, this->blobs_[0]->mutable_gpu_data());
    int m = bottom[0]->count()/this->channels_;
    Dtype bias_correction_factor = m > 1 ? Dtype(m)/(m-1) : 1;
    caffe_gpu_axpby(this->variance_.count(), bias_correction_factor,
        this->variance_.gpu_data(), this->moving_average_fraction_,
        this->blobs_[1]->mutable_gpu_data());
  }

  // normalize variance
  caffe_gpu_add_scalar(this->variance_.count(), this->eps_, this->variance_.mutable_gpu_data());
  caffe_gpu_powx(this->variance_.count(), this->variance_.gpu_data(), Dtype(0.5),
      this->variance_.mutable_gpu_data());


  if (this->phase_ == TEST) { /////////Quantize/////////
      // normalize variance
      caffe_gpu_add_scalar(this->variance_.count(), this->eps_, this->weights_quantized_variance_[0]->mutable_gpu_data());
      caffe_gpu_powx(this->variance_.count(), this->weights_quantized_variance_[0]->gpu_data(), Dtype(0.5),
          this->weights_quantized_variance_[0]->mutable_gpu_data());
    int rounding = this->phase_ == TEST ? this->rounding_ : QuantizationParameter_Rounding_STOCHASTIC;
    this->QuantizeWeights_gpu(this->weights_quantized_variance_, rounding, false);
    // replicate variance to input size
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
        this->batch_sum_multiplier_.gpu_data(), this->weights_quantized_variance_[0]->gpu_data(), 0.,
        this->num_by_chans_.mutable_gpu_data());
  }else{
      // replicate variance to input size
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
          this->batch_sum_multiplier_.gpu_data(), this->variance_.gpu_data(), 0.,
          this->num_by_chans_.mutable_gpu_data());
  }
  
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->channels_ * num,
      spatial_dim, 1, 1., this->num_by_chans_.gpu_data(),
      this->spatial_sum_multiplier_.gpu_data(), 0., this->temp_.mutable_gpu_data());
  caffe_gpu_div(this->temp_.count(), top_data, this->temp_.gpu_data(), top_data);
  // TODO(cdoersch): The caching is only needed because later in-place layers
  //                 might clobber the data.  Can we skip this if they won't?
  caffe_copy(this->x_norm_.count(), top_data,
      this->x_norm_.mutable_gpu_data());
	  
  if (this->phase_ == TEST) { /////////Quantize/////////
    //std::cout << "bn_out_quant befor: " << top[0]->cpu_data()[1] << " " << top[0]->cpu_data()[2] << std::endl;
    this->QuantizeLayerOutputs_gpu(top[0]->mutable_gpu_data(),top[0]->count());///////Quantize///////
    //std::cout << "bn_out_quant after: " << top[0]->cpu_data()[1] << " " << top[0]->cpu_data()[2] << std::endl;
  }
}

template <typename Dtype>
void BatchNormRistrettoLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff;
  if (bottom[0] != top[0]) {
    top_diff = top[0]->gpu_diff();
  } else {
    caffe_copy(this->x_norm_.count(), top[0]->gpu_diff(), this->x_norm_.mutable_gpu_diff());
    top_diff = this->x_norm_.gpu_diff();
  }
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  if (this->use_global_stats_) {
    caffe_gpu_div(this->temp_.count(), top_diff, this->temp_.gpu_data(), bottom_diff);
    return;
  }
  const Dtype* top_data = this->x_norm_.gpu_data();
  int num = bottom[0]->shape()[0];
  int spatial_dim = bottom[0]->count()/(this->channels_*bottom[0]->shape(0));
  // if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
  //
  // dE(Y)/dX =
  //   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
  //     ./ sqrt(var(X) + eps)
  //
  // where \cdot and ./ are hadamard product and elementwise division,
  // respectively, dE/dY is the top diff, and mean/var/sum are all computed
  // along all dimensions except the channels dimension.  In the above
  // equation, the operations allow for expansion (i.e. broadcast) along all
  // dimensions except the channels dimension where required.

  // sum(dE/dY \cdot Y)
  caffe_gpu_mul(this->temp_.count(), top_data, top_diff, bottom_diff);
  caffe_gpu_gemv<Dtype>(CblasNoTrans, this->channels_ * num, spatial_dim, 1.,
      bottom_diff, this->spatial_sum_multiplier_.gpu_data(), 0.,
      this->num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemv<Dtype>(CblasTrans, num, this->channels_, 1.,
      this->num_by_chans_.gpu_data(), this->batch_sum_multiplier_.gpu_data(), 0.,
      this->mean_.mutable_gpu_data());

  // reshape (broadcast) the above
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
      this->batch_sum_multiplier_.gpu_data(), this->mean_.gpu_data(), 0.,
      this->num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, this->channels_ * num,
      spatial_dim, 1, 1., this->num_by_chans_.gpu_data(),
      this->spatial_sum_multiplier_.gpu_data(), 0., bottom_diff);

  // sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_mul(this->temp_.count(), top_data, bottom_diff, bottom_diff);

  // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_gemv<Dtype>(CblasNoTrans, this->channels_ * num, spatial_dim, 1.,
      top_diff, this->spatial_sum_multiplier_.gpu_data(), 0.,
      this->num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemv<Dtype>(CblasTrans, num, this->channels_, 1.,
      this->num_by_chans_.gpu_data(), this->batch_sum_multiplier_.gpu_data(), 0.,
      this->mean_.mutable_gpu_data());
  // reshape (broadcast) the above to make
  // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, this->channels_, 1, 1,
      this->batch_sum_multiplier_.gpu_data(), this->mean_.gpu_data(), 0.,
      this->num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num * this->channels_,
      spatial_dim, 1, 1., this->num_by_chans_.gpu_data(),
      this->spatial_sum_multiplier_.gpu_data(), 1., bottom_diff);

  // dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y
  caffe_gpu_axpby(this->temp_.count(), Dtype(1), top_diff,
      Dtype(-1. / (num * spatial_dim)), bottom_diff);

  // note: this->temp_ still contains sqrt(var(X)+eps), computed during the forward
  // pass.
  caffe_gpu_div(this->temp_.count(), bottom_diff, this->temp_.gpu_data(), bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(BatchNormRistrettoLayer);


}  // namespace caffe
